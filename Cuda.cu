#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
#include <random>
 
using namespace std;
using namespace chrono;
 
void setup(int* array, int arraySize, int numObstacles) {
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<int> dis(0, arraySize - 1);
 
    for (int i = 0; i < numObstacles; ++i) {
        int row = dis(gen);
        int col = dis(gen);
        array[row * arraySize + col] = 1;
    }
}
 
__global__ void findObstaclesCUDA(int* array, int arraySize, int* result) {
    int obstacleCount = 0;
 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
 
    for (int i = idx; i < arraySize * arraySize; i += stride) {
        int row = i / arraySize;
        int col = i % arraySize;
 
        if (array[row * arraySize + col] == 1) {
            int index = atomicAdd(&obstacleCount, 1);
            result[index * 2] = row;
            result[index * 2 + 1] = col;
        }
    }
}
 
int main() {
    int arraySize = 256;
    int numObstacles = 100;
 
    int* array = new int[arraySize * arraySize];
    hipMallocManaged(&array, arraySize * arraySize * sizeof(int));
 
    auto startCUDA = high_resolution_clock::now();
 
    setup(array, arraySize, numObstacles);
 
    int* resultCUDA;
    hipMallocManaged(&resultCUDA, numObstacles * 2 * sizeof(int));
 
    int threadsPerBlock = 256;
    int blocksPerGrid = (arraySize * arraySize + threadsPerBlock - 1) / threadsPerBlock;
    findObstaclesCUDA<<<blocksPerGrid, threadsPerBlock>>>(array, arraySize, resultCUDA);
    hipDeviceSynchronize();
 
    hipError_t hipError_t;
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        cout << "CUDA error: " << hipGetErrorString(hipError_t) << endl;
    }
 
    auto stopCUDA = high_resolution_clock::now();
    auto durationCUDA = duration_cast<milliseconds>(stopCUDA - startCUDA);
 
    cout << "CUDA C/C++ Execution Time: " << durationCUDA.count() << " ms\n";
 
    delete[] array;
    hipFree(resultCUDA);
 
    return 0;
}
